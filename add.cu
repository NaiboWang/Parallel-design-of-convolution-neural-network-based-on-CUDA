#include "hip/hip_runtime.h"
#include "add.cuh"
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

hipError_t initCuda()
{
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}
	printf("Cuda init success!\n");
	return cudaStatus;
}
static void HandleError(hipError_t err,const char *file,int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
	}
}
hipError_t convWithCuda(float* src, float* dst, float* filter, int imageOutSize, int imageInSize, int filterSize)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, filterSize * filterSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, imageInSize *imageInSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, imageOutSize *imageOutSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, src, imageInSize *imageInSize * sizeof(float),hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!a");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_c, filter, filterSize *  filterSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!c");
		goto Error;
	}
	dim3 grid(1);
	dim3 block(imageOutSize, imageOutSize);
	// Launch a kernel on the GPU with one thread for each element.
	conv2MexCuda << <grid,block >> > (dev_a, dev_b, dev_c,imageOutSize,imageInSize,filterSize);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	HANDLE_ERROR(cudaStatus =hipDeviceSynchronize());
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dst, dev_b, imageOutSize *imageOutSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!b");
		goto Error;
	}
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return cudaStatus;
}
__global__ void conv2MexCuda(float* src, float* dst, float* filter, int imageOutSize, int imageInSize, int filterSize)
{
	int row = threadIdx.x;
	if (row < 0 || row > imageOutSize - 1)
		return;
	int col = threadIdx.y;
	if (col < 0 || col > imageOutSize - 1)
		return;
	int dstIndex = col * imageOutSize + row;
	int fSize = filterSize * filterSize;
	dst[dstIndex] = 0;
#pragma unroll
	for (int fy = 0; fy < filterSize; fy++) {
#pragma unroll
		for (int fx = 0; fx < filterSize; fx++) {
			float filterItem = filter[--fSize];
			float imageItem = src[row + fx + (fy + col)*imageInSize];
			dst[dstIndex] += filterItem*imageItem;
		}
	}
}
